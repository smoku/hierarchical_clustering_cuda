#include "hip/hip_runtime.h"
#include <math.h>
#include <limits.h>
#include <stdio.h>
#include <cutil_inline.h>
#include <cudpp/cudpp.h>

// Liczba punktów
const int n = 5;

// Liczba wymiarów
const int d = 2;

__global__ void calculate_distances(float* d_vectors, float* d_distance) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;

  int index = i + j*n;
  d_distance[index] = 0;

  // sprawdzenie czy nie przekraczamy "n" oraz rozpatrujemy tylko dolną część tablicy dwuwymiarowej
  if (i<n && j<n && i<j) {  
    for (int k=0; k<d; k++) {
      // obliczenie odległości euklidesowej (bez pierwiastka - nie potrzebny do obliczeń)
      float r = d_vectors[i*d + k] - d_vectors[j*d + k];
      d_distance[index] += r*r;
    }
  }
}

__global__ void merge_clusters(float* d_distance, int* d_dendrogram, int* d_merged_clusters, int step) {
  int x = blockIdx.x*blockDim.x+threadIdx.x;
  int y = blockIdx.y*blockDim.y+threadIdx.y;
  
  if (x == 0 && y == 0) {
    // znajdź najmniejszą wartość w macierzy odległości
    int min_index = INT_MAX;
    for (int i=0; i<n-1; i++) {
      for (int j=i+1; j<n; j++) {
        // wez pod uwagę jeżeli klaster "i" i klaster "j" nie są złączone
        if (!d_merged_clusters[i] && !d_merged_clusters[j]) {
          int index = i + j*n;
          if (min_index == INT_MAX || d_distance[index] < d_distance[min_index])
            min_index = index;
        }
      }
    }
    
    if (min_index != INT_MAX) {
      int i = min_index/n;
      int j = min_index%n;
  
      // zapisz połączone klastry
      d_dendrogram[step] = i;
      d_dendrogram[step+(n-1)] = j;
      
      // wyłącz jeden z klastrów z dalszym obliczeń
      d_merged_clusters[j] = 1;
    }
  }
}

__global__ void update_distances(float* d_distance, int* d_dendrogram, int* d_merged_clusters, int step) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;

  int index = i + j*n;

  // sprawdzenie czy nie przekraczamy "n" oraz rozpatrujemy tylko dolną część tablicy dwuwymiarowej
  if (i<n && j<n && i<j) {
    // aktualizujemy tablice tylko dla klastrów, które nie są scalone
    if (!d_merged_clusters[i] && !d_merged_clusters[j]) {
      int candidate_index = INT_MAX;

      if (d_dendrogram[step] == i)
        candidate_index = d_dendrogram[step+(n-1)] + j*n;
      else if (d_dendrogram[step] == j)
        candidate_index = d_dendrogram[step+(n-1)] + i*n;
      
      if (candidate_index != INT_MAX && d_distance[candidate_index] < d_distance[index])
        d_distance[index] = d_distance[candidate_index];
      
    }
  }
}

void print_step_results(int step, float* h_distance, int* h_dendrogram, int* h_merged_clusters) {
  printf("\n\n\n");    
  printf("Krok %i", step+1);
  printf("\n\n");
  
  printf("Macierz odległości:");
  for (int i=0; i<n; i++) {
    printf("\n");
    for (int j=0; j<n; j++) {
      if (!h_merged_clusters[i] && !h_merged_clusters[j])
        printf(" %f ",h_distance[i*n+j]);
      else
        printf("     M     ");
    }
  }
  printf("\n\n");

  printf("Klastry złączone:");
  printf("\n");
  for (int i=0;i<n;i++) {
    printf(" C%i ",i);
  }
  printf("\n");
  for (int i=0;i<n;i++) {
    printf(" %i ",h_merged_clusters[i]);
  }
  printf("\n\n");

  printf("Dendrogram:");
  for (int i=0;i<(n-1)*2;i++) {
    if (i%(n-1)==0)
      printf("\n");
    printf(" %i ",h_dendrogram[i]);
  }
  printf("\n\n");
}


int main(int argc, char** argv) {
  if (cutCheckCmdLineFlag(argc, (const char**)argv, "device"))
    cutilDeviceInit(argc, argv);
  else
    hipSetDevice(cutGetMaxGflopsDeviceId());

  float* h_vectors=(float*)malloc(sizeof(float)* n*d);
  float* h_distance=(float*)malloc(sizeof(float)* n*n);
  int* h_dendrogram=(int*)malloc(sizeof(int)* (n-1)*2);
  int* h_merged_clusters=(int*)malloc(sizeof(int)* n);  

  float* d_vectors;
  cutilSafeCall(hipMalloc((void**)&d_vectors,sizeof(float)* n*d));
  
  float* d_distance;
  cutilSafeCall(hipMalloc((void**)&d_distance,sizeof(float)* n*n));
  
  int* d_dendrogram;
  cutilSafeCall(hipMalloc((void**)&d_dendrogram,sizeof(int)* (n-1)*2));
  
  int* d_merged_clusters;
  cutilSafeCall(hipMalloc((void**)&d_merged_clusters,sizeof(int)* n));

  // wypełnij wektory losowymi wartościami
  // for (int i=0;i<n*d;i++) {
  //   h_vectors[i] = rand()%10;
  // }
  
  // P0
  h_vectors[0*d] = -5;
  h_vectors[0*d+1] = 4;
  
  // P1
  h_vectors[1*d] = 4;
  h_vectors[1*d+1] = -3;
  
  // P2
  h_vectors[2*d] = 5;
  h_vectors[2*d+1] = -5;
  
  // P3
  h_vectors[3*d] = -3;
  h_vectors[3*d+1] = 5;
  
  // P4
  h_vectors[4*d] = 1;
  h_vectors[4*d+1] = 1;
  
  // wypełnij tablice dendrogramu zerami
  for (int i=0;i<(n-1)*2;i++) {
    h_dendrogram[i] = 0;
  }
  
  for (int i=0;i<n;i++) {
    h_merged_clusters[i] = 0;
  }
  
  printf("Punkty wejściowe:");
  for (int i=0;i<n*d;i++) {
    if (i%d==0) {
      printf("\n");
      printf("P%i:", i/d);
    }
    printf(" %f ",h_vectors[i]);
  }
  printf("\n\n\n");
  
  // skopiuj tablice punktów z hosta do karty graficznej
  cutilSafeCall(hipMemcpy(d_vectors,h_vectors,n*d*sizeof(float),hipMemcpyHostToDevice));
  
  // skopiuj tablice dendrogramu z hosta do karty graficznej
  cutilSafeCall(hipMemcpy(d_dendrogram,h_dendrogram,(n-1)*2*sizeof(int),hipMemcpyHostToDevice));
  
  // skopiuj tablice scalonych klastrów z hosta do karty graficznej
  cutilSafeCall(hipMemcpy(d_merged_clusters,h_merged_clusters,n*sizeof(int),hipMemcpyHostToDevice));

  // calculate grid width and height
  int width=n/16+(((n%16)!=0)?1:0);
  int height=n/16+(((n%16)!=0)?1:0);

  dim3 grid(width,height);
  dim3 block(16,16);
  
  dim3 sgrid(1,1);
  dim3 sblock(4,4);

  // oblicz macierz odległości
  calculate_distances<<<grid,block>>>(d_vectors, d_distance);
    
  // skopiuj macierz odległości do hosta
  cutilSafeCall(hipMemcpy(h_distance,d_distance,n*n*sizeof(float), hipMemcpyDeviceToHost));
  
  printf("Macierz odległości:");
  for (int i=0;i<n*n;i++) {
    if (i%n==0)
      printf("\n");
    printf(" %f ",h_distance[i]);
  }
  printf("\n\n");
  
  cutilSafeCall(hipMemcpy(d_dendrogram,h_dendrogram,(n-1)*2*sizeof(int),hipMemcpyHostToDevice));
  
  for (int step=0; step < n-1; step++){
    
    // scal klastry dla najmniejszej odległości
    merge_clusters<<<sgrid,sblock>>>(d_distance, d_dendrogram, d_merged_clusters, step);
    
    // zaktualizuj macierz odległości z uwzględnieniem scalonych klastrów
    update_distances<<<grid,block>>>(d_distance, d_dendrogram, d_merged_clusters, step);
    
    // prześlij obliczenia na host (tylko w celu wyświetlenia obliczeń dla kroku)
    cutilSafeCall(hipMemcpy(h_distance,d_distance,n*n*sizeof(float), hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(h_dendrogram,d_dendrogram,(n-1)*2*sizeof(int), hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(h_merged_clusters,d_merged_clusters,n*sizeof(int), hipMemcpyDeviceToHost));
    
    // wyświetl wyniki dla danego kroku
    print_step_results(step, h_distance, h_dendrogram, h_merged_clusters);
    
  }
  
  hipDeviceReset();
}